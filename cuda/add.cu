#include <iostream>
#include <hip/hip_runtime.h>
#include <sys/time.h>
using namespace std ;
const int SPIN = 1LL<<19 ;
using ll = long long ;
static double start ;
int n ;
double walltime() {
   struct timeval tv ;
   gettimeofday(&tv, 0) ;
   return tv.tv_sec + 0.000001 * tv.tv_usec ;
}
double duration() {
   double now = walltime() ;
   double r = now - start ;
   start = now ;
   return r ;
}
__global__ void add(int *a, int *b, int *c, int n)
{
    int index = blockDim.x * blockIdx.x + threadIdx.x;
    if (index < n) {
       int x = a[index] ;
       int y = b[index] ;
       int smcnt = 0 ;
       for (int i=0; i<SPIN; i++) {
          if (y != 0) {
             x -= (y + (y - (y < 0 ? 16 : 0)) / 17) >> 1 ;
             y += (7*x)+(x>>1);
             smcnt++ ;
          }
       }
       a[index] = x ;
       b[index] = y ;
       c[index] = smcnt ;
    }
}
int *a, *b, *c, *s ;
ll *cnt, totwork ;
void getwork(int i, int &nxtval) {
   a[i] = nxtval;
   b[i] = 7*a[i] + (a[i]>>1);
   c[i] = 0;
   s[i] = a[i];
   cnt[i] = 1;
   nxtval-- ;
}
int main(int argc, char *argv[]) {
    double firsttime = walltime() ;
    int asize = atol(argv[1]) ;
    int bsize = atol(argv[2]) ;
    n = asize * bsize ;
    int *d_a, *d_b, *d_c;
    int size = n * sizeof(int);
    int nxtval = -11000000 ;
    duration() ;
    a = (int *)malloc(size);
    b = (int *)malloc(size);
    c = (int *)malloc(size);
    s = (int *)malloc(size);
    cnt = (ll *)malloc(size * 2);
    hipMalloc((void **)&d_a, size);
    hipMalloc((void **)&d_b, size);
    hipMalloc((void **)&d_c, size);
    cout << "Alloced in " << duration() << endl ;
    for (int i = 0; i < n; i++)
       getwork(i, nxtval);
    cout << "Starting . . ." << endl ;
    for (ll iter=1; ; iter++) {
       hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
       hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);
       cout << "Copied in " << duration() << endl ;
       add<<<asize, bsize>>>(d_a, d_b, d_c, n);
       cout << "Executed in " << duration() << endl ;
       hipMemcpy(a, d_a, size, hipMemcpyDeviceToHost);
       hipMemcpy(b, d_b, size, hipMemcpyDeviceToHost);
       hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);
       cout << "Copied back in " << duration() << endl ;
       for (int i=0; i<n; i++) {
	  if (c[i] == 0) {
             cout << "No work at " << i << endl ;
	     exit(10) ;
          }
          cnt[i] += c[i];
	  if (b[i] == 0) {
	     cout << s[i] << " " << cnt[i] << " " << a[i] << endl ;
	     totwork += cnt[i] ;
	     getwork(i, nxtval);
	  }
       }
       double rate = iter * (double)SPIN * n / (walltime() - firsttime) ;
       cout << "Reviewed in " << duration() << " rate " << rate << endl ;
    }
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    free(a);
    free(b);
    free(c);
    cout << "Freed in " << duration() << endl ;
    return 0;
}
